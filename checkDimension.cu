#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void checkIndex()
{
    printf("threadIdx:(%d, %d, %d) blockIdx(%d, %d, %d) blockDim(%d, %d, %d)""gridDim:(%d, %d, %d)\n", 
            threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z,
            blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
}

int main ()
{
    int nElem = 6;
    dim3 block(3);  //each block contain 3 threads
    dim3 grid ((nElem+block.x-1)/block.x);  //block.x means the total number of blocks in the first dim

    printf("grid.x %d, grid.y %d, grid.z %d\n",grid.x,grid.y,grid.z);
    printf("block.x %d, block.y %d, block.z %d\n",block.x, block.y, block.z);

    checkIndex <<<grid, block>>>();

    hipDeviceReset();
    return 0;
}