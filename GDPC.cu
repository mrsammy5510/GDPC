#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <fstream>
#include <algorithm>    
#include <math.h>
#include <stdio.h>
#include <sstream>
#include <iomanip> // Header file needed to use setprecision
#include <numeric> // for using iota
#include "./common/common.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#include <hip/hip_runtime.h>

using namespace std;

ifstream dataset;
ofstream ids_left_file("ids_left.txt");
ofstream ids_right_file("ids_right.txt");
ofstream ids_file("ids.txt");
ofstream cluster_result_file("cluster_result.txt");
ofstream dist_tmp_file("dist_tmp.txt");
ofstream VP_tree_file("VP_tree.txt");
ofstream leaf_file("leaf.txt");
ofstream tmp_file("tmp.txt");


typedef struct VP_node
{
    int vp = -1;
    float rad = -1;
}VP_node;

class leaf_node
{
    public:
        int id[32] = {};
        leaf_node()
        {
            memset(id, -1, 32 * sizeof(int));
        }
};


template<typename T> class d_vector
{
private:
    T* begin;
    T* end;

    int length;
    int capacity;
public:
    __device__ __host__ d_vector(): length(0), capacity(16)
    {
        begin = new T [capacity];
        end = begin - 1;
    }
    __device__ __host__ ~d_vector()
    {
        delete[] begin;
        begin = nullptr;
    }
    __device__ void expand()
    {
        capacity *= 2;
        T* tmp_begin = new T[capacity];
        memcpy(tmp_begin, begin, sizeof(T) * length);
        delete[] begin;
        begin = tmp_begin;
        end = begin + length - 1;
    }
    __device__ void push_back(T data)
    {
        if(length >= capacity)
        {
            expand();
        }

        end++;
        length++;

        *end = data;
    }
    __device__ T pop_back()
    {
        T end_element = *end;
        end--;
        length--;
        return end_element;
    }
    __device__ int size()
    {
        return length;
    }
    __device__ T pos(int position)
    {
        return *(begin + position);
    }
};


vector<vector<float>> readcsv(string filename)
{
    ifstream file(filename);

    vector<vector<float>> data; // 用二維vector來儲存表格數據

    if (file) {
        string line;
        while (getline(file, line)) { // 逐行讀取檔案
            vector<float> row;
            stringstream ss(line);
            string field;
            while (getline(ss, field, ',')) { // 以逗號分隔字段
                row.push_back(stof(field)); // 將字段轉換為float並加入vector中
            }
            data.push_back(row); // 將這一行數據加入到二維vector中
        }

    } else {
        cout << "Error: failed to open file" << endl;
    }

    return data;
}

inline float dist(vector<float> point1, vector<float> point2)
{
    float sum = 0;
    for(int i = 0;i<point1.size();i++)
    {
        sum+= pow(point1[i]-point2[i],2);
    }
    return sqrt(sum);
}


inline void normalize(vector<vector<float>> &datapoints)
{
    vector<float> attributes;
    for(int j = 0; j<datapoints[0].size(); j++)     //j defines which attribute is modifying now
    {
        for(int i = 0; i<datapoints.size(); i++)    //i is which point is modifying now
        {
            attributes.push_back(datapoints[i][j]);
        }
        auto max = *max_element(attributes.begin(), attributes.end());
        auto min = *min_element(attributes.begin(), attributes.end());
        for(int i = 0; i<datapoints.size(); i++)
        {
            datapoints[i][j] = (datapoints[i][j]-min)/(max-min);
        }
    }
}

inline void flatten(thrust::host_vector<float> &_1d_datapoints, vector<vector<float>> &datapoints)
{
    for(int row = 0; row < datapoints.size(); row++)
    {
        for(int col = 0; col< datapoints[0].size(); col++)
        {
            _1d_datapoints.push_back(datapoints[row][col]);
        }
    }
}

inline void decide_first_VP(vector<vector<float>> &datapoints, vector<VP_node> &VP, int data_size)
{
    vector<float> first_VP_cand;
    VP_node *first_VP = new VP_node;

    for(int i = 0; i < data_size; i++)
    {
        first_VP_cand.push_back(dist(datapoints[0], datapoints[i]));
    }
    auto max = max_element(first_VP_cand.begin(), first_VP_cand.end());
    first_VP->vp = max - first_VP_cand.begin();
    VP[0] = (*first_VP);
}



__device__ __host__ inline float dist(float* point1, float* point2, int dim)
{
    float sum = 0;
    for(int i = 0;i< dim;i++)
    {
        sum+= pow((point1[i]-point2[i]),2);
    }
    return sqrt(sum);
}

__global__ void get_dist(float* datapoints, int dim, int* ids, VP_node* VP, int VP_id, float* dist_tmp, int size)
{
    
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < size)
    {
            dist_tmp[i] = dist(&datapoints[VP[VP_id].vp * dim], &datapoints[ids[i] * dim], dim);
    }
}


void quickSort(float* arr, int left, int right, int* ids) {
    int i = left, j = right;
    float pivot = arr[(left + right) / 2];

    /* partition */
    while (i <= j) {
        while (arr[i] < pivot)
            i++;
        while (arr[j] > pivot)
            j--;
        if (i <= j) {
            swap(arr[i],arr[j]);
            swap(ids[i],ids[j]);
            i++;
            j--;
        }
    };

    /* recursion */
    if (left < j)
        quickSort(arr, left, j, ids);
    if (i < right)
        quickSort(arr, i, right, ids);
}


void recur_build_VP_tree(int dim, int* ids, int ids_size, 
                        VP_node* VP, int VP_id, leaf_node* leaf, int height, int total_size, float* d_datapoints)
{
    int size = ids_size;    //Total points in this subtree
    float* dist_tmp = new float[size];
    int mid = size/2;       //The medium
    

    //For CUDA computing
    dim3 block (32);
    dim3 grid ((size + block.x - 1)/block.x);
    int* d_ids;
    VP_node* d_VP;
    float* d_dist_tmp;


    //------------------------------------------------------------------------------------------------

    if(size>32)
    {
        
        hipMalloc((void** )&d_ids, size * sizeof(int));
        hipMalloc((void** )&d_VP, (int)pow(2, height) * sizeof(VP_node));
        hipMalloc((void** )&d_dist_tmp, size * sizeof(float));
        
        hipMemcpy(d_ids, ids, size * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_VP, VP, (int)pow(2, height) * sizeof(VP_node), hipMemcpyHostToDevice);
        
        
        get_dist<<<grid, block>>>(d_datapoints, dim, d_ids, d_VP, VP_id, d_dist_tmp, size);
        hipDeviceSynchronize();


        thrust::sort_by_key(thrust::device, d_dist_tmp, d_dist_tmp + size, d_ids);
        hipMemcpy(dist_tmp, d_dist_tmp, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(ids, d_ids, size * sizeof(int), hipMemcpyDeviceToHost);
        
        /*
        hipFree(d_ids);
        hipFree(d_VP);
        hipFree(d_dist_tmp);
        */
        
        VP[VP_id].rad = dist_tmp[mid];
        int* ids_left = (int* )malloc(sizeof(int) * mid);
        int* ids_right = (int* )malloc(sizeof(int) * (size - mid));

        memcpy(ids_left, ids, mid * sizeof(int));
        memcpy(ids_right, ids + mid, (size - mid) * sizeof(int));
        //Recursively building left subtree and right subtree
        if(mid>32)
        {
            VP[2*VP_id+1].vp = ids[mid-1];
            VP[2*VP_id+2].vp = ids[size-1];
        }
        

        recur_build_VP_tree(dim,  ids_left,        mid, VP, 2 * VP_id + 1, leaf, height, total_size, d_datapoints);
        recur_build_VP_tree(dim, ids_right, size - mid, VP, 2 * VP_id + 2, leaf, height, total_size, d_datapoints);
        //---------------------------------------------------------------------------------------------
        
    }
    else
    {
        memcpy(leaf[VP_id-(int)pow(2, height)+1].id, ids, size * sizeof(int)); 
    }
    

    
}
void initialize(float* datapoints, vector<vector<float>> datapoints_vector, int dim)
{
    for(int i = 0; i < datapoints_vector.size(); i++)
    {
        for(int j = 0; j < datapoints_vector[0].size(); j++)
        {
            datapoints[i * dim + j] = datapoints_vector[i][j];
        }
    }
}

inline void decide_first_VP(vector<vector<float>> &datapoints, VP_node* VP, int data_size)
{
    vector<float> first_VP_cand;
    VP_node *first_VP = new VP_node;

    for(int i = 0; i < data_size; i++)
    {
        first_VP_cand.push_back(dist(datapoints[0], datapoints[i]));
    }
    auto max = max_element(first_VP_cand.begin(), first_VP_cand.end());
    first_VP->vp = max - first_VP_cand.begin();
    VP[0] = (*first_VP);
}

void print_VP_tree(VP_node* VP, leaf_node* leaf, int VP_size, int leaf_size)
{
    for(int i = 0; i < VP_size; i++)
    {
        VP_tree_file<<"VP["<<i<<"]: "<<VP[i].vp<<"\t"<<"rad: "<<VP[i].rad<<endl;
    }
    for(int j = 0; j < leaf_size; j++)
    {
        leaf_file<<"leaf "<<j<<": ";
        int k = 0;
        while(leaf[j].id[k] != -1)
        {
            leaf_file<<leaf[j].id[k]<<" ";
            k++;
        }
        leaf_file<<endl;
    }
}

__global__ void GDPC_rho(float* d_datapoints, VP_node* d_VP, leaf_node* d_leaf, int VP_size, int dim, float dc, int* d_rho,
                        d_vector<int>* cover_leaves)
{   
    unsigned int pid = blockIdx.x * blockDim.x + threadIdx.x;

    cover_leaves[pid] = *(new d_vector<int>);

    d_vector<int> stack;
    stack.push_back(0);

    while(stack.size()!=0)
    {

        int vp_now = stack.pop_back();
        if(vp_now >= VP_size)
        {           
            int leaf_tmp = 0;
            while(d_leaf[vp_now - VP_size].id[leaf_tmp] != -1 && leaf_tmp <= 31)
            {
                cover_leaves[pid].push_back(d_leaf[vp_now - VP_size].id[leaf_tmp]);
                leaf_tmp++;
            }
        }
        else
        {
            if(dist(&d_datapoints[d_VP[vp_now].vp * dim], &d_datapoints[pid * dim], dim) - dc <= d_VP[vp_now].rad)
            {
                stack.push_back(2 * vp_now + 1);
            }
            if(dist(&d_datapoints[d_VP[vp_now].vp * dim], &d_datapoints[pid * dim], dim) + dc >= d_VP[vp_now].rad)
            {
                stack.push_back(2 * vp_now + 2);
            }
            
        }
    }
    
    for(int i = 0; i < cover_leaves[pid].size(); i++)
    {   
        if(dist(&d_datapoints[cover_leaves[pid].pos(i) * dim], &d_datapoints[pid * dim], dim) <= dc)
        {
            d_rho[pid]++;
        }
    }

    /*
    unsigned int pid = blockIdx.x * blockDim.x + threadIdx.x;
    d_vector<int> cover_leaves;
    d_vector<int> stack;
    stack.push_back(0);

    while(stack.size()!=0)
    {

        int vp_now = stack.pop_back();
        if(vp_now >= VP_size)
        {           
            int leaf_tmp = 0;
            while(d_leaf[vp_now - VP_size].id[leaf_tmp] != -1 && leaf_tmp <= 31)
            {
                cover_leaves.push_back(d_leaf[vp_now - VP_size].id[leaf_tmp]);
                leaf_tmp++;
            }
        }
        else
        {
            if(dist(&d_datapoints[d_VP[vp_now].vp * dim], &d_datapoints[pid * dim], dim) - dc <= d_VP[vp_now].rad)
            {
                stack.push_back(2 * vp_now + 1);
            }
            if(dist(&d_datapoints[d_VP[vp_now].vp * dim], &d_datapoints[pid * dim], dim) + dc >= d_VP[vp_now].rad)
            {
                stack.push_back(2 * vp_now + 2);
            }
            
        }
    }
    
    for(int i = 0; i < cover_leaves.size(); i++)
    {   
        if(dist(&d_datapoints[cover_leaves.pos(i) * dim], &d_datapoints[pid * dim], dim) <= dc)
        {
            d_rho[pid]++;
        }
    }
    */
}

__global__ void GDPC_result()
{

}

int main()
{

    vector<vector<float>> datapoints_vector;
    datapoints_vector = readcsv("./csv file/S2.csv");
    //datapoints_vector = readcsv("./csv file/3D_spatial_network.csv");
    //datapoints_vector = readcsv("./GDPC/csv file/S2.csv");
    //datapoints_vector = readcsv("./csv file/3D_spatial_network.csv");
    normalize(datapoints_vector);
    int data_size = datapoints_vector.size();
    int dim = datapoints_vector[0].size();
    
    float dc = 0.00969449058;
    
    
    //flatten the datapoints into 1D
    float* datapoints = new float[data_size * dim];
    initialize(datapoints, datapoints_vector, dim);
    //----------------------------------------------------------------------


    //For VP tree
    int h = ceil(log(data_size/32)/log(2));     //tree height (h-1)
    VP_node* VP = new VP_node[(int)pow(2, h) - 1];
    decide_first_VP(datapoints_vector, VP, data_size);
    int ids [data_size];
    for(int i = 0; i < data_size; i++){
        ids[i] = i;
    }
    leaf_node leaf[(int)pow(2, h)] = {};
    

    //-----------------------------------------------------------------
    hipDeviceReset();  //Clean up the device
    hipDeviceSynchronize();    // Warm up

    float* d_datapoints;
    hipMalloc((void** )&d_datapoints, data_size * dim * sizeof(float));
    hipMemcpy(d_datapoints, datapoints, data_size * dim * sizeof(float), hipMemcpyHostToDevice);

    timespec t_building_VP_tree, tmp;

    clock_gettime(CLOCK_MONOTONIC, &t_building_VP_tree);
    
    recur_build_VP_tree(dim, ids, data_size, VP, 0, leaf, h, data_size, d_datapoints);

    clock_gettime(CLOCK_MONOTONIC, &tmp);
    cout << "time passed for building VP_tree parallelly is: " << 
    (tmp.tv_sec - t_building_VP_tree.tv_sec)*1000 + 
    (tmp.tv_nsec - t_building_VP_tree.tv_nsec)/1000000 << "ms" << endl;
    
    print_VP_tree(VP, leaf, (int)pow(2, h) - 1, (int)pow(2, h));

    dim3 block (32);
    dim3 grid ((data_size + block.x - 1)/block.x); 
    int* d_rho;
    VP_node* d_VP;
    leaf_node* d_leaf;
    hipMalloc((void**)&d_rho, data_size * sizeof(int));
    hipMalloc((void**)&d_VP, ((int)pow(2, h) - 1) * sizeof(VP_node));
    hipMalloc((void**)&d_leaf, (int)pow(2, h) * sizeof(leaf_node));

    hipMemset(d_rho, -1, data_size * sizeof(int));
    hipMemcpy(d_VP, VP, ((int)pow(2, h) - 1) * sizeof(VP_node), hipMemcpyHostToDevice);
    hipMemcpy(d_leaf, leaf, (int)pow(2, h) * sizeof(leaf_node), hipMemcpyHostToDevice);

    d_vector<int>* cover_leaves;
    hipMalloc((void**)&cover_leaves, data_size * sizeof(d_vector<int>)); 

    GDPC_rho<<<grid, block>>>(d_datapoints, d_VP, d_leaf, (int)pow(2, h) - 1, dim, dc, d_rho, cover_leaves);
    hipDeviceSynchronize();
    GDPC_result<<<grid, block>>();


    int* rho;
    rho = (int*)malloc(sizeof(int) * data_size);
    hipMemcpy(rho, d_rho, sizeof(int) * data_size, hipMemcpyDeviceToHost);

    
    for(int i = 0; i <data_size; i++)
    {
        //printf("rho %d = %d\t", i, rho[i]);
    }
    
     
    return 0;
}